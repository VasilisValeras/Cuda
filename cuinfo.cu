// Vasileios Valeras 4031


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/* 
 * Retrieves and prints information for every installed NVIDIA
 * GPU device
 */
void cuinfo_print_devinfo()
{
    int num_devs, i;
    hipDeviceProp_t dev_prop;
    
    hipGetDeviceCount(&num_devs);
    if (num_devs == 0)
    {
        printf("No CUDA devices found.\n");
        return;
    }
    
    for (i = 0; i < num_devs; i++)
    {
        hipGetDeviceProperties(&dev_prop, i);
        
        printf("Device %d:\n", i);
        printf("\tName: %s\n", dev_prop.name);
        printf("\tCUDA Version: %d.%d\n", dev_prop.major, dev_prop.minor);
        printf("\tCompute Capability: %d.%d\n", dev_prop.major, dev_prop.minor);
        printf("\tStreaming Multiprocessors: %d\n", dev_prop.multiProcessorCount);
        printf("\tCUDA Cores: %d\n", 
            (dev_prop.major == 2 && dev_prop.minor == 1) ? dev_prop.multiProcessorCount * 48 :
            (dev_prop.major == 3) ? dev_prop.multiProcessorCount * 192 :
            (dev_prop.major == 5 && dev_prop.minor == 2) ? dev_prop.multiProcessorCount * 128 :
            (dev_prop.major == 5 && dev_prop.minor == 3) ? dev_prop.multiProcessorCount * 192 :
            (dev_prop.major == 6 && dev_prop.minor == 0) ? dev_prop.multiProcessorCount * 128 :
            (dev_prop.major == 6 && dev_prop.minor == 1) ? dev_prop.multiProcessorCount * 128 :
            (dev_prop.major == 7 && dev_prop.minor == 0) ? dev_prop.multiProcessorCount * 64 :
            dev_prop.multiProcessorCount * 128); // assuming architecture of compute capability 8.x and above
        printf("\tMax Threads per Block: %d\n", dev_prop.maxThreadsPerBlock);
        printf("\tTotal Global Memory: %lu bytes\n", dev_prop.totalGlobalMem);
        printf("\tShared Memory per Block: %lu bytes\n", dev_prop.sharedMemPerBlock);
    }
}


int main()
{
	cuinfo_print_devinfo();
	return 0;
}
